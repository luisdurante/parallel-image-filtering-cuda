#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "Image.h"
#include "stb_image.h"
#include "stb_image_write.h"
#include <stdio.h>
#include <cstdio>
#include <stdint.h>
#include <stddef.h>
#include <cmath>
#include <string>

hipError_t addWithCuda(uint8_t *originalData, unsigned int size, char filter);

__global__ void addKernelGray(uint8_t* originalData, unsigned int size, int totalThreads)
{
    int threadNum = threadIdx.x;
    int finished = 0;
    int pixelIndex = threadNum;

    while (finished == 0)
    {
        if (pixelIndex > size - 3) {
            break;
        }

        int gray = (originalData[pixelIndex] + originalData[pixelIndex+1] + originalData[pixelIndex+2]) / 3; 
        originalData[pixelIndex] = gray;
        pixelIndex += totalThreads;
    }
    


    //for (size_t i = 0; i < size; i++)
    //{
        
    //}
    
    /*else if (threadNum == 1) {
        for (size_t i = 0; i < size; i+= 3)
        {
            // https://www.techrepublic.com/blog/how-do-i/how-do-i-convert-images-to-grayscale-and-sepia-tone-using-c/
            uint8_t inputRed = originalData[i]; 
            uint8_t inputGreen = originalData[i+1];
            uint8_t inputBlue = originalData[i+2];

            int red = (inputRed * 0.393) + (inputGreen * 0.769) + (inputBlue * 0.189); 
            int green = (inputRed * 0.349) + (inputGreen * 0.686) + (inputBlue * 0.168);
            int blue = (inputRed * 0.272) + (inputGreen * 0.534) + (inputBlue * 0.131);

            if(red > 255) red = 255;
            if(red < 0 ) red = 0;
            if(green > 255) green = 255;
            if(green < 0 ) green = 0;
            if(blue > 255) blue = 255;
            if(blue < 0 ) blue = 0;

            result_sepia[i] = red;
            result_sepia[i+1] = green;
            result_sepia[i+2] = blue;
        }
    } else if (threadNum == 2) {
        for (size_t i = 0; i < size; i+= 3)
        {
            originalData[i] = 255 - originalData[i];
            originalData[i+1] = 255 - originalData[i+1];
            originalData[i+2] = 255 - originalData[i+2];
        }
    }*/

}

__global__ void addKernelSepia(uint8_t* originalData, unsigned int size, int totalThreads)
{
    int threadNum = threadIdx.x;
    int finished = 0;
    int pixelIndex = threadNum;
    
    while (finished == 0)
    {
        if (pixelIndex > 0 && pixelIndex % 3 != 0) {
            pixelIndex += totalThreads;
            continue;
        }

        if (pixelIndex > size - 3) {
            break;
        }

        // https://www.techrepublic.com/blog/how-do-i/how-do-i-convert-images-to-grayscale-and-sepia-tone-using-c/
        uint8_t inputRed = originalData[pixelIndex]; 
        uint8_t inputGreen = originalData[pixelIndex+1];
        uint8_t inputBlue = originalData[pixelIndex+2];

        int red = (inputRed * 0.393) + (inputGreen * 0.769) + (inputBlue * 0.189); 
        int green = (inputRed * 0.349) + (inputGreen * 0.686) + (inputBlue * 0.168);
        int blue = (inputRed * 0.272) + (inputGreen * 0.534) + (inputBlue * 0.131);

        if(red > 255) red = 255;
        if(red < 0 ) red = 0;
        if(green > 255) green = 255;
        if(green < 0 ) green = 0;
        if(blue > 255) blue = 255;
        if(blue < 0 ) blue = 0;

        originalData[pixelIndex] = red;
        originalData[pixelIndex+1] = green;
        originalData[pixelIndex+2] = blue;
        
        pixelIndex += totalThreads;
    }
}

__global__ void addKernelInverted(uint8_t* originalData, unsigned int size, int totalThreads)
{
    int threadNum = threadIdx.x;
    int finished = 0;
    int pixelIndex = threadNum;

    while (finished == 0)
    {
        if (pixelIndex > 0 && pixelIndex % 3 != 0) {
            pixelIndex += totalThreads;
            continue;
        }

        if (pixelIndex > size - 3) {
            break;
        }

        originalData[pixelIndex] = 255 - originalData[pixelIndex];
        originalData[pixelIndex+1] = 255 - originalData[pixelIndex+1];
        originalData[pixelIndex+2] = 255 - originalData[pixelIndex+2];
        
        pixelIndex += totalThreads;
    }
}


class MyException : public std::exception
{
    private:
       std::string ex;
    public:
        MyException(const char* err) : ex(err) {}       
};

Image::Image(const char* fileName) {
    if(read(fileName)) {
        printf("Lendo %s\n", fileName);
        size = w * h * channels;
    } else {
        printf("Falha na leitura %s\n", fileName);
        throw MyException("Exception");
    }
}

Image::Image(int w, int h, int channels) : w(w), h(h), channels(channels){
    size = w * h * channels;
    data = new uint8_t[size];
}

Image::Image(const Image& img) : Image(img.w,img.h,img.channels) {
    memcpy(data, img.data, img.size);
}

Image::~Image() {
    stbi_image_free(data);
}

bool Image::read(const char* fileName) {
    data = stbi_load(fileName, &w, &h, &channels, 0);
    return data != NULL;
}

bool Image::write(const char* fileName) {
    ImageType type = getFileType(fileName);
	int success;
    switch (type) {
        case PNG:
            success = stbi_write_png(fileName, w, h, channels, data, w * channels);
        break;
        case BMP:
            success = stbi_write_bmp(fileName, w, h, channels, data);
        break;
        case JPG:
            success = stbi_write_jpg(fileName, w, h, channels, data, 100);
        break;
        case TGA:
            success = stbi_write_tga(fileName, w, h, channels, data);
        break;
    }
    return success != 0;
}

ImageType Image::getFileType(const char* fileName) {
    const char* ext = strrchr(fileName, '.');
	if(ext != nullptr) {
		if(strcmp(ext, ".png") == 0) {
			return PNG;
		}
		else if(strcmp(ext, ".jpg") == 0) {
			return JPG;
		}
		else if(strcmp(ext, ".bmp") == 0) {
			return BMP;
		}
		else if(strcmp(ext, ".tga") == 0) {
			return TGA;
		}
	}
	return PNG;
}

Image& Image::grayscale_lum() {
    // preserva a luminosidade
    if (channels < 3) {
        throw MyException("Exception");
    } else {
        addWithCuda(data, size, 'g');
    }
    return *this;
}

Image& Image::sepia() {
    if (channels < 3) {
        throw MyException("Exception");
    } else {
        addWithCuda(data, size, 's');
    }

    return *this;
}

Image& Image::invertColors() {
    if (channels < 3) {
        throw MyException("Exception");
    } else {
        addWithCuda(data, size, 'i');
    }
    return *this;
}

hipError_t addWithCuda(uint8_t *originalData, unsigned int size, char filter)
{
    uint8_t *dev_originalData = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    
    cudaStatus = hipMalloc((void**)&dev_originalData, size * sizeof(uint8_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_originalData, originalData, size * sizeof(uint8_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
  
    // Launch a kernel on the GPU with one thread for each element.
    if (filter == 'g')
        addKernelGray<<<1, 1024>>>(dev_originalData,  size, 1024);
    else if (filter == 's')
        addKernelSepia<<<1, 1024>>>(dev_originalData, size, 1024);
    else
        addKernelInverted<<<1, 1024>>>(dev_originalData, size, 1024);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(originalData, dev_originalData, size * sizeof(uint8_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! passo 9");
        goto Error;
    }

Error:
    hipFree(dev_originalData);
    
    return cudaStatus;
}